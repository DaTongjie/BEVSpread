// Copyright (c) Megvii Inc. All rights reserved.

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK 256
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

__global__ void voxel_pooling_forward_kernel(int batch_size, int num_points, int num_channels, int num_voxel_x,
                                             int num_voxel_y, int num_voxel_z, const int *geom_xyz,
                                             const float *input_features, float *output_features, int *pos_memo) {
  // Each thread process only one channel of one voxel.
  int blk_idx = blockIdx.x;
  int thd_idx = threadIdx.x;
  int idx = blk_idx * blockDim.x + thd_idx;
  if (idx >= batch_size * num_points * num_channels) {
    return;
  } else {
    int batch_idx = idx / (num_points * num_channels);
    int point_idx = idx / num_channels;
    int channel_idx = idx % num_channels;
    int x = geom_xyz[point_idx * 3];
    int y = geom_xyz[point_idx * 3 + 1];
    int z = geom_xyz[point_idx * 3 + 2];
    // if coord of current voxel is out of boundary, return.
    if (x < 0 || x >= num_voxel_x || y < 0 || y >= num_voxel_y || z < 0 || z >= num_voxel_z) {
      return;
    }
    pos_memo[point_idx * 3] = batch_idx;
    pos_memo[point_idx * 3 + 1] = y;
    pos_memo[point_idx * 3 + 2] = x;
    atomicAdd(
          &output_features[(batch_idx * num_voxel_y * num_voxel_x + y * num_voxel_x + x) * num_channels + channel_idx],
          input_features[point_idx * num_channels + channel_idx]);
  }
}

void voxel_pooling_forward_kernel_launcher(int batch_size, int num_points, int num_channels, int num_voxel_x,
                                           int num_voxel_y, int num_voxel_z, const int *geom_xyz,
                                           const float *input_features, float *output_features, int *pos_memo,
                                           hipStream_t stream) {
  hipError_t err;

  dim3 blocks(DIVUP(batch_size * num_points * num_channels, THREADS_PER_BLOCK)); // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK);

  voxel_pooling_forward_kernel<<<blocks, threads, 0, stream>>>(batch_size, num_points, num_channels, num_voxel_x,
                                                               num_voxel_y, num_voxel_z, geom_xyz, input_features,
                                                               output_features, pos_memo);
  // cudaDeviceSynchronize();  // for using printf in kernel function
  err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
  }
}
